#include "hip/hip_runtime.h"
#include <math.h>
#include <string.h>
#include <stdio.h>
#include "timer.h"


// define mesh size, number of blocks NBL and threads per block TPB
#define NN 4096
#define NM 4096
#define NBLX 32
#define NBLY 32
#define TPBX 16
#define TPBY 16


// -------------------------------------------------------
// CUDA device function that performs jacobi method
// -------------------------------------------------------
__global__ void jacobi(long int *Anew, long int *A, int maxrow, int maxcol){

    size_t rowinit = threadIdx.x + blockDim.x * blockIdx.x;
    size_t colinit = threadIdx.y + blockDim.y * blockIdx.y;
    size_t rowstride = gridDim.x * blockDim.x;
    size_t colstride = gridDim.y * blockDim.y;

    int iter = 0;
    int iter_max = 1000;
    long int tol = 1.0e-6;
    long int error = 1.0;

    // until our error is what we want or we have done too many iterations, keep recalculating A matrix and error
    while ( error > tol && iter < iter_max ){
        error = 0.0;
        // for loop to calculate the new A matrix and the error
        // here we use our variables based on threadIdx, blockDim, blockIdx, and gridDim
        for (size_t j = rowinit; j < NN; j += rowstride) {
            for (size_t i = colinit; i < NM; i += colstride) {
                Anew[j*NM +i] = 0.25 * ( A[j*NM + i+1] + A[j*NM + i-1] + A[(j-1)*NM +i] + A[(j+1)*NM +i]);
                error = max( error, abs(Anew[j*NM +i] - A[j*NM +i]));
            }
        }
        // Synchronize threads - make sure all threads are done before continuing
        __syncthreads();    

        // for loop to set A matrix equal to the new A matrix
        // here we use our variables based on threadIdx, blockDim, blockIdx, and gridDim
        for (size_t j = rowinit; j < NM; j += rowstride) {
            for (size_t i = colinit; i < NM; i += colstride) {
                A[j*NM +i] = Anew[j*NM +i];    
            }
        }

        // Synchronize threads - make sure all threads are done before continuing
        __syncthreads();
    }

}


// ------------
// main program
// ------------
int main(void){
    // variables
    int n = NN;
    int m = NM;
    long int *A;
    long int *Anew;
    size_t size = n * m * sizeof(long int);

    // allocate host memory
    A = (long int *) malloc(size);

    // initialize matrix
    for (int j = 0; j < n; j++){
        for(int i=0; i<m; i++){
            A[j*m +i] = 1.0;
        }
    }
    printf("Jacobi relaxation Calculation: %d x %d mesh\n", n, m);
    // allocate device memory
    hipMalloc((void **)&Anew, size);

    // copy input data to device
    hipMemcpy(Anew, A, size, hipMemcpyHostToDevice);

    StartTimer();

    // Apply jacobi by launching a sufficient number of blocks
    printf("\n---------------------------\n");
    printf("Launching jacobi kernel\n");
    printf("---------------------------\n");
    printf("Matrix elements   = %d x %d = %d\n",NN,NM,NN*NM);
    printf("Blocks            = %d x %d = %d\n",NBLX,NBLY,NBLX*NBLY);
    printf("Threads per block = %d x %d = %d\n",TPBX,TPBY,TPBX*TPBY);
    printf("Kernel copies     = %d\n",NBLX*NBLY*TPBX*TPBY);

    jacobi<<<dim3(NBLX,NBLY),dim3(TPBX,TPBY)>>>(Anew, A, n, m);

    // copy results back to host
    hipMemcpy(Anew, A, size, hipMemcpyDeviceToHost);
  
    // deallocate memory
    hipFree(Anew);

    double runtime = GetTimer();
    printf(" total: %f s\n", runtime / 1000);
    return 0;
}
